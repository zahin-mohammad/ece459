#include "hip/hip_runtime.h"
#include <stdio.h>


#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

// Use this as the random seed in the pseudorandom function (see instructions)
#define SEED 20210418
#define NUM_VOTERS 100000
#define NUM_SIMULATIONS 500000
#define VOTE_BLOCK_INCREMENT 3125 // 100k / 32
// Pseudo-random number generator. This provides consistent reproducible results so that
// you can test your program.
__device__ float pseudorandom( uint x ) {
    uint value = x;
    value = (value ^ 61) ^ (value>>16);
    value *= 9;
    value ^= value << 4;
    value *= 0x27d4eb2d;
    value ^= value >> 15;
    return (float) value / (float) INT_MAX;
}

extern "C" __global__ void Vote(
    const float input[NUM_VOTERS][3], 
    uint out[NUM_SIMULATIONS][2]) 
{
    // Threads Per Block: 32
    // Num Blocks : Dx = 800, Dy = 625
    int simulation_number = blockIdx.x + blockIdx.y*800;
    // printf("%d\n",simulation_number );
    int vote_block = threadIdx.x;

    int vote_a = 0;
    int vote_b = 0;

    for (int j = 0; j < VOTE_BLOCK_INCREMENT; j ++) {
        int i = vote_block*VOTE_BLOCK_INCREMENT +j;
        double voter_x = input[i][0];
        double voter_y = input[i][1];

        double p = pseudorandom(SEED+simulation_number+i);
        // Should remove these if's
        if (p < voter_x) {
            vote_a += 1;
        } else if (voter_x <= p && p < voter_x + voter_y) {
            vote_b += 1;
        }
    }
    atomicAdd(&out[simulation_number][0], vote_a);
    atomicAdd(&out[simulation_number][1], vote_b);
}
