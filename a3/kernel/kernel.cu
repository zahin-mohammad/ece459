#include "hip/hip_runtime.h"
// Very minimal skeleton for the kernel

#include <stdio.h>

// Values copied from cnn.rs
#define INPUT_DIM 100
#define FILTER_DIM 5 // should be factor of INPUT_DIM
#define CONV_OUT_DIM  (INPUT_DIM / FILTER_DIM)
#define CONV_LAYER_SIZE 10
#define OUT_NEURON_DIM  (CONV_OUT_DIM * CONV_OUT_DIM * CONV_LAYER_SIZE)
#define OUT_LAYER_SIZE 10

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

// out assumed to be initialized to 0
extern "C" __global__ void OutputLayer(
    const double input[OUT_NEURON_DIM], 
    const double output_layer[OUT_LAYER_SIZE][OUT_NEURON_DIM], 
    double out[OUT_LAYER_SIZE]) 
{
    int out_index = blockIdx.x;
    int layer_index = threadIdx.x;
    double prod = 0.0;
    #pragma unroll
    for (int j= 0; j<125; j++){
        int i = layer_index*125 + j; 
        prod += input[i]*output_layer[out_index][i];
    }
    atomicAdd(&out[out_index], prod);
}

// out assumed to be initialized to 0
extern "C" __global__ void ConvolutionLayerAndReLU(
    const double input[INPUT_DIM][INPUT_DIM], 
    const double conv_layer[CONV_LAYER_SIZE][FILTER_DIM][FILTER_DIM], 
    double out[CONV_LAYER_SIZE][CONV_OUT_DIM][CONV_OUT_DIM]) 
{
    int filter_index = blockIdx.x;
    int filter_row_index = threadIdx.x;
    int filter_col_index = threadIdx.y;

    double prod = 0.0;
    #pragma unroll
    for (int y = 0; y < FILTER_DIM; y++)
        #pragma unroll
        for (int x = 0; x < FILTER_DIM; x++)
            prod += input[filter_row_index*FILTER_DIM + y][filter_col_index*FILTER_DIM + x] * conv_layer[filter_index][y][x];

    // ReLU
    prod = fmax(prod, 0.0);
    out[filter_index][filter_row_index][filter_col_index] = prod;
}